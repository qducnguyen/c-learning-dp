//Tính tổng của 2 mảng dùng chương trình song song CUDA 
#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#define N 128
#define Nr_Thread_Points 4

// Kernel
__global__ void KernelTinhTong2Mang(int *a, int *b, int *c) 
{	int i,index, start, stop;
	index = blockIdx.x * blockDim.x + threadIdx.x;
    start = index*Nr_Thread_Points;
    stop  = start + Nr_Thread_Points;
    for (i=start;i<stop;i++)
		*(c + i) = *(a + i) + *(b + i);
}

int main(void) { 
// Host code
  	int i,*a_cpu, *b_cpu, *c_cpu; 
  	a_cpu = (int *) malloc (N*sizeof(int));
	b_cpu = (int *) malloc (N*sizeof(int));
	c_cpu = (int *) malloc (N*sizeof(int));
  	for (i = 0; i < N; i++) {
    	*(a_cpu+i) = i * 2;
    	*(b_cpu+i) = i * 3;
  }
//Khai báo biến trên GPU
	int *a_gpu, *b_gpu, *c_gpu;
	hipMalloc((void**)&a_gpu,N*sizeof(int));
	hipMalloc((void**)&b_gpu,N*sizeof(int));
	hipMalloc((void**)&c_gpu,N*sizeof(int));
//Copy Input CPU -> GPU
	hipMemcpy(a_gpu, a_cpu, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b_cpu, N*sizeof(int), hipMemcpyHostToDevice);
//Define structure: Thread, block
    int SoLuongBlocks, SoLuongThreads;
    SoLuongBlocks = 4;
    SoLuongThreads = N/(SoLuongBlocks*Nr_Thread_Points); 
	dim3 dimBlock(SoLuongThreads);
	dim3 dimGrid(SoLuongBlocks);
//Invoke Kernel
	KernelTinhTong2Mang<<<dimGrid,dimBlock>>>(a_gpu,b_gpu,c_gpu);
//Copy Output GPU -> CPU
	hipMemcpy(c_cpu, c_gpu, N*sizeof(int), hipMemcpyDeviceToHost);
//Giải phóng bộ nhớ trên GPU
	hipFree(a_gpu);hipFree(b_gpu);hipFree(c_gpu);
// In KQ
  for (i = 0; i < N; i++) printf("%d  ",*(c_cpu+i));
  printf("\n");
return 0;
}

#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

#define M 16	// row
#define N 32	// col
#define GridSizeX 2 
#define GridSizeY 2
#define BlockSizeX 2
#define BlockSizeY 4
#define ThreadSizeX M / (GridSizeX * BlockSizeX) 
#define ThreadSizeY N / (GridSizeY * BlockSizeY) 

// Kernel
__global__ void SumOfTwoMatricesKernel(int *A, int *B, int *C) 
{	
	int indexX, indexY;
	indexX = threadIdx.x + blockIdx.x*blockDim.x ;
	indexY = threadIdx.y + blockIdx.y*blockDim.y;

	int startX = indexX * ThreadSizeX;
  int stopX  = startX + ThreadSizeX;
  int startY = indexY * ThreadSizeY;
  int stopY  = startY + ThreadSizeY;

  for (int i = startX; i < stopX; i++){
  	for (int j = startY; j < stopY;j++){
  		*(C + i*N+j) = *(A + i*N+j)  + *(B + i*N+j); 
  	}
  }
}

int main(void) { 
// Host code
  int i, j, *A_cpu, *B_cpu, *C_cpu; 
  // Declaration on CPU 
  A_cpu = (int *) malloc (N*M*sizeof(int));
	B_cpu = (int *) malloc (N*M*sizeof(int));
	C_cpu = (int *) malloc (N*M*sizeof(int));
		// Declaration
  	for (i = 0; i < M; i++) {
  		for (j = 0; j < N;j++){
  			 	*(A_cpu+i*N+j) = (i*N+j) * 2;
    			*(B_cpu+i*N+j) = (i*N+j) * 3;
  		}
  }
	//	Declaration on GPU
	int *A_gpu, *B_gpu, *C_gpu;
	hipMalloc((void**)&A_gpu,N*M*sizeof(int));
	hipMalloc((void**)&B_gpu,N*M*sizeof(int));
	hipMalloc((void**)&C_gpu,N*M*sizeof(int));
	
	//Copy Input CPU -> GPU
	hipMemcpy(A_gpu, A_cpu, N*M*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B_cpu, N*M*sizeof(int), hipMemcpyHostToDevice);
	
	//Define structure: Thread, block
	dim3 dimGrid(GridSizeX, GridSizeY);
	dim3 dimBlock(BlockSizeX, BlockSizeY);

	//Invoke Kernel
	SumOfTwoMatricesKernel<<<dimGrid,dimBlock>>>(A_gpu,B_gpu,C_gpu);

	//Copy Output GPU -> CPU
	hipMemcpy(C_cpu, C_gpu, N*M*sizeof(int), hipMemcpyDeviceToHost);
		for (i = 0; i < M; i++) {
  		for (j = 0; j < N;j++){
  			  printf("%d ", *(C_cpu+i*N+j));
  		}
  		printf("\n");
  	}

	//Free Memomry
	hipFree(A_gpu);hipFree(B_gpu);hipFree(C_gpu);
	free(A_cpu); free(B_cpu);free(C_cpu);
}

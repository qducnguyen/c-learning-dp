#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#define M       32
#define GridSize 4
#define BlockSize M/GridSize
//
__global__ void reduce0(int *g_idata, int *g_odata){
  extern __shared__ int sdata[M];
// each thread loads one element from global to shared mem
	int tid, i, s;
  tid =  threadIdx.x;
	i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = *(g_idata+i);
  __syncthreads();
// do reduction in shared mem
  for(s=1; s < blockDim.x; s *= 2) {
    if(tid%(2*s) == 0){
    sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
// write result for this block to global mem
  if (tid==0) *(g_odata + blockIdx.x) = sdata[0];
}
//=========================
int main(){
        int i, Sumcpu = 0;
        int *Ricpu, *Rocpu;
        Ricpu  = (int *) malloc (M*sizeof(int));
        Rocpu  = (int *) malloc (GridSize*sizeof(int));
        for (i=0; i<M; i++) {*(Ricpu+i) = i+1; Sumcpu = Sumcpu + *(Ricpu+i);}
        printf("Sum by CPU: %d \n", Sumcpu);
     // CUDA code
    //1. Delare and Allocate Mem on GPU
         int *Rigpu,*Rogpu;
         hipMalloc((void**)&Rigpu ,M*sizeof(int));
         hipMalloc((void**)&Rogpu ,GridSize*sizeof(int));
    //2. Copy Input from CPU to GPU
         hipMemcpy(Rigpu,Ricpu,M*sizeof(int),hipMemcpyHostToDevice);
    //3. Define Block and Thread Structure
         dim3 dimGrid(GridSize);
         dim3 dimBlock(BlockSize);
         reduce0<<<dimGrid,dimBlock>>>(Rigpu,Rogpu);
    //5. Copy Output from GPU to CPU
         hipMemcpy(Rocpu,Rogpu,GridSize*sizeof(int),hipMemcpyDeviceToHost);
         printf("Sum by GPU: \n");
         for (i = 0;i < GridSize;i++ ) printf("%d \n",*(Rocpu+i));
    //6. Free Mem on CPU and GPU
         free(Ricpu);free(Rocpu);
         hipFree(Rigpu);hipFree(Rigpu);
return 0;
}

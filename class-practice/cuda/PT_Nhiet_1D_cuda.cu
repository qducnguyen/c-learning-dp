#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#define  M       32
#define  Ntime   100
#define  dt      0.01
#define  dx      0.1
#define  D       0.1

#define GridSize 2
#define BlockSize 8
#define ThreadSize M/(GridSize*BlockSize)

//=========================
void KhoiTao(float *T){

  int i;
  for (  i = 0 ; i < M ; i++ )
     *(T+i) = 25.0;
}
//=========================
__global__ void DHB2(float *T, float *dT){

    float c,l,r;
	int i,index, start, stop;
	index = blockIdx.x * blockDim.x + threadIdx.x;
    start = index*ThreadSize;
    stop  = start + ThreadSize;     
    for (i=start;i<stop;i++) {
      c = *(T+i);
      l = (i==0)   ? 100.0 : *(T+(i-1));
      r = (i==M-1) ? 25.0  : *(T+(i+1));
      *(dT+i) = D*(r-2*c+l)/(dx*dx);
      __syncthreads();
    }
}
__global__ void ODE(float *T,float *dT) 
{
	int i,index, start, stop;
	index = blockIdx.x * blockDim.x + threadIdx.x;
    start = index*ThreadSize;
    stop  = start + ThreadSize;
    for (i=start;i<stop;i++)
      *(T+i) = *(T+i) + dt*(*(dT+i));
	__syncthreads();
}
//=========================
int main(){

    int i,t;
    float *Tcpu,*dTcpu;
    Tcpu  = (float *) malloc ((M)*sizeof(float));
    dTcpu = (float *) malloc ((M)*sizeof(float));
    KhoiTao(Tcpu);
    
    // CUDA code

    //1. Delare and Allocate Mem on GPU
    float *Tgpu,*dTgpu;
    hipMalloc((void**)&Tgpu ,M*sizeof(int));
    hipMalloc((void**)&dTgpu,M*sizeof(int));

    //2. Copy Input from CPU to GPU
    hipMemcpy(Tgpu,Tcpu,M*sizeof(int),hipMemcpyHostToDevice); 

    //3. Define Block and Thread Structure
    dim3 dimGrid(GridSize);
    dim3 dimBlock(BlockSize);

    //4. Execute
    for (t=0;t<Ntime;t++) {
        DHB2<<<dimGrid,dimBlock>>>(Tgpu,dTgpu);
        ODE<<<dimGrid,dimBlock>>>(Tgpu,dTgpu);
     }

    //5. Copy Output from GPU to CPU
    hipMemcpy(Tcpu,Tgpu,M*sizeof(int),hipMemcpyDeviceToHost);
    for (i = 0;i < M;i++ ) printf("%f \n",*(Tcpu+i));

    //6. Free Mem on CPU and GPU
    free(Tcpu);free(dTcpu);
    hipFree(Tgpu);hipFree(dTgpu);
    return 0;
}
